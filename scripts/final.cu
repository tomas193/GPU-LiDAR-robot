
//Paralelo 


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <ctime>
#include <cstdlib>
#include <fstream>
#include <string>
#include <sstream>
#include <cmath>

using namespace std;

vector<float> vec_lec;
vector<vector<int>> visited;

int lectures(const vector<int>& actual) {
    cout<<"nodo actual: "<<actual[0]<<","<<actual[1]<<endl;
   
    system("python3 read.py");

    // Abre el archivo en modo de lectura
    ifstream archivo("lectures.txt");

    // Verifica si se pudo abrir el archivo correctamente
    if (!archivo.is_open()) {
        cerr << "No se pudo abrir el archivo." << endl;
        return 1;
    }
    string linea; //variable que guarda cada linea de texto
    getline(archivo,linea); //obtener linea de archivo
    stringstream ss(linea);
    string valor;

    while (getline(ss,valor,',')){
       float entero=stof(valor); //convertir string a float
       vec_lec.push_back(entero);} //insetar datos en vec_lec

   // Cierra el archivo después de leerlo
    archivo.close();

    float grados=0.01745329252;
    int frame=30; //centimetros
    int k=90; //angulo donde inicia la medicion  
    
     
   for (const auto& p:vec_lec){
      vector<int> aux; //guardar x,y
      double hip=((p*100)/frame);//convertir medicion en espacio de estados
      int x=round(hip*(cos(k*grados)));//componente x de la medicion
      int y=round(hip*(sin(k*grados)));//componente y de la medicion
      x+=actual[0];y+=actual[1];
      aux.push_back(x);aux.push_back(y);
      bool flag=false; //flag for repeated coord

      for (int i=0;i<visited.size();i++){
      if (aux==visited[i]){flag=true;break;}}//checar si una coordenada se repite
      if (flag==false && aux!=actual){visited.push_back(aux);}//insertar si no esta en la lista
      k+=1;}  

      
  for (int i=0;i<visited.size();i++){
   cout<<visited[i][0]<<","<<visited[i][1]<<endl;}
   cout<<"no permitidos: "<<visited.size()<<endl;	 

}

const int n = 501;     //10001; 

const int inicio_x = 10;
const int inicio_y = 10;
const int goal_x = 0; //10000;
const int goal_y = 0; //10000;

static void checking(hipError_t err){
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);}}

__host__ __device__ 
int get_index(int x, int y) {
    if(x <= n && y <n) return y * n + x;

    else return -0;}

__host__ __device__ 
void get_coordinates(int index, int* x, int* y) {
    *y = index / n;
    *x = index % n;}

void construir_mundo(int* mundo) {
    // Inicializar todo el mundo a 0
    for (int i = 0; i < n*n; i++) {
        mundo[i] = 0;}

    // Marcar el estado inicial como visitado
    int index = get_index(inicio_x, inicio_y);
    mundo[index] = 1;
    
   //mundo[get_index(goal_x, goal_y)] = 2;
}

__device__
int movimientos[8][2] = {
    {1, 0}, // Derecha
    {-1, 0}, // Izquierda
    {0, 1}, // Arriba
    {0, -1}, // Abajo
    {1, 1}, // Arriba-Derecha
    {-1, 1}, // Arriba-Izquierda
    {-1, -1}, // Abajo-Izquierda
    {1, -1} // Abajo-Derecha
};

__global__
void kernel(int* ruta, int inicio, int final, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx != inicio && idx < n){
      int x, y;
      get_coordinates(idx, &x, &y);

      int inicio_x, inicio_y;
      get_coordinates(inicio, &inicio_x, &inicio_y);

      int min_dist = INT_MAX;
      int nearest_idx = -1;

      for (int i = 0; i < 8; i++) {
          int vecino_x = x + movimientos[i][0];
          int vecino_y = y + movimientos[i][1];
          
          if (vecino_x >= 0 && vecino_x < n && vecino_y >= 0 && vecino_y < n) {
              int vecino_idx = get_index(vecino_x, vecino_y);
              int dist = abs(inicio_x - vecino_x) + abs(inicio_y - vecino_y);

              if (dist < min_dist) {
                  min_dist = dist;
                  nearest_idx = vecino_idx;
              }
          }
      }
         ruta[idx] = nearest_idx;
      }
    }
  


int main() {
    int size = n*n*sizeof(int);
    unsigned t0, t1;
    int inicio = get_index(inicio_x, inicio_y);
    int final =  get_index(goal_x, goal_y);
    int *h_mundo; int* d_mundo;
    int* h_ruta; int* d_ruta;

    vector<int> actual={inicio_x,inicio_y};
  
    lectures(actual);
    
    hipError_t err;
    err = hipMalloc((void **) &d_mundo, size);
    checking(err);
    err = hipMalloc((void **) &d_ruta, size); 
    checking(err);

    h_mundo = (int*) malloc(size);
    h_ruta = (int*) malloc(size);

    construir_mundo(h_mundo); 
    
    h_ruta[inicio] = -1; // El nodo inicial no tiene un nodo anterior

    hipMemcpy(d_mundo, h_mundo, size, hipMemcpyHostToDevice);
    hipMemcpy(d_ruta, h_ruta, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int numBlocks = (n*n + threadsPerBlock - 1) / threadsPerBlock;
    
    t0 = clock();
    kernel<<<numBlocks, threadsPerBlock>>>(d_ruta, inicio, final, n*n);
    hipDeviceSynchronize();
    t1 = clock();

    hipMemcpy(h_ruta, d_ruta, size, hipMemcpyDeviceToHost);

    // Inicializar el nodo actual al nodo final
    int nodo_actual = final;

    // Mientras no hayamos llegado al nodo inicial
    vector<std::pair<int, int>> trayectoria;

    while (nodo_actual != inicio) {
        
      int x, y;
      get_coordinates(nodo_actual, &x, &y);
      //printf("Nodo: (%d, %d)\n", x, y);

      trayectoria.push_back(std::make_pair(x, y));

      // Mueve al nodo actual hacia atrás a lo largo de la ruta
      nodo_actual = h_ruta[nodo_actual];}
      
      // Finalmente, imprimir el nodo inicial
      reverse(trayectoria.begin(), trayectoria.end());

    for (const auto& p: trayectoria) {
    cout << "(" << p.first << ", " << p.second << ") ";}
    cout << std::endl;
    double time = (double(t1-t0)/CLOCKS_PER_SEC);
    cout << "Execution Time: " << time << endl;
}
    
